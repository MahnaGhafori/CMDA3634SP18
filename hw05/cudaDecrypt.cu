#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include "functions.c"

__global__ void find(unsigned int p, unsigned int g, unsigned int h, unsigned int x)
{
unsigned int threadID = threadId.x;
unsigned int blockID = blockID.x;
unsigned int nBlocks = blockDim.x;
unsigned int curr = threadID + nBlocks*blockID;

if (modExp(g, curr, p) == h)
{
x = curr;
}

}

int main (int argc, char **argv) {

  /* Q4 Make the search for the secret key parallel on the GPU using CUDA. */

//declare storage for an ElGamal cryptosytem
  unsigned int n, p, g, h, x;
  unsigned int Nints;

  //get the secret key from the user
  printf("Enter the secret key (0 if unknown): "); fflush(stdout);
  char stat = scanf("%u",&x);

  printf("Reading file.\n");

  /* Q3 Complete this function. Read in the public key data from public_key.txt
    and the cyphertexts from messages.txt. */
  FILE *pk = fopen("public_key.txt", "r");
  FILE *mes = fopen("public_key.txt", "r");

  fscanf(pk, "%d %d %d %d", n, p, g, h);
  fscanf(mes, "%d", Nints);

// where the secret key will be stored when it's found
unsigned int *sk;
hipMalloc(%sk, sizeof(unsigned int));

// how the find method will iterate through
int Nthreads = 32; 
int Nblocks = (p + Nthreads - 2)/Nthreads;

  // find the secret key
  if (x==0 || modExp(g,x,p)!=h) {
    printf("Finding the secret key...\n");
    double startTime = clock();
   find<<<Nthreads, Nblocks>>>(p, g, h, sk);
   hipMemcpy(x, sk, sizeof(unsigned int), hipMemcpyDeviceToHost);
hipDeviceSynchronize();
    double endTime = clock();
    double totalTime = (endTime-startTime)/CLOCKS_PER_SEC;
    double work = (double) p;
    double throughput = work/totalTime;

    printf("Searching all keys took %g seconds, throughput was %g values tested per second.\n", totalTime, throughput);
  }
hipFree(sk);

  /* Q3 After finding the secret key, decrypt the message */
  int y;
  unsigned char *foundMes = (unsigned char *) malloc(Nints * sizeof(unsigned char)); // will contain the message once it is decrypted
  unsigned int c1, c2; // will select each ciphertext pair
  
  for (y = 0; y < Nints; y++)
    {
      fscanf(mes, "%d %d", c1, c2);
      foundMes[y] = ( pow(c1, p-2) * c2 ) % mod p; // convert back to 
    }

  fclose(pk);
  fclose(mes);

  return 0;

  return 0;
}
